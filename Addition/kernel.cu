#include "hip/hip_runtime.h"
#include<iostream>
#include "common/book.h"

__global__ void add(int a, int b, int* c) {
	*c = a + b;
}

int main(void) {
	int c;
	//host variable
	int* dev_c;
	//device variable

	const char* file = __FILE__;
	int line = __LINE__;
	hipError_t err;

	err = hipMalloc((void**)&dev_c, sizeof(int));

	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}

	add<<<1,1>>>(2, 7, dev_c);

	err = hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}

	printf("\n 2 + 7 =  %d", c);
	hipFree(dev_c);
	return 0;
}